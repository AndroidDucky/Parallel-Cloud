#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ctype.h"
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "cuFunctions.h"

__device__ int globalIdx = 0;
__device__ int* mutex;
__device__ void lock(){
    while(atomicCAS(mutex, 0, 1)!=0);
}
__device__ void unlock(){
    atomicExch(mutex,0);
}


__device__ int IsLetter(char c){
    if(c == 'a'|| c == 'b'||c == 'c'||c == 'd'||c == 'e'||c == 'f'||c == 'g'||c == 'h'||c == 'i'||c == 'j'||c == 'k'||c == 'l'||c == 'm'||c == 'n'||c == 'o'||c == 'p'||c == 'q'||c == 'r'||c == 's'||c == 't'||c =='u'||c == 'v'||c == 'w'||c == 'x'||c == 'y'||c == 'z'||c == 'A'|| c == 'B'||c == 'C'||c == 'D'||c == 'E'||c == 'F'||c== 'G'||c == 'H'||c == 'I'||c == 'J'||c == 'K'||c == 'L'||c == 'M'||c == 'N'||c == 'O'||c == 'P'||c == 'Q'||c == 'R'||c =='S'||c == 'T'||c == 'U'||c == 'V'||c == 'W'||c == 'X'||c == 'Y'||c == 'Z'){
        return 1;
    }
    return 0;
}

__device__ char * my_strcpy(char *dest, const char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != 0);
  return dest;
}

__global__ void parselineKernel( char **linesIn, char** arrOut, const int linesLength){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    //const int y = blockIdx.y * blockDim.y + Threa     ^~~~dIdx.y;

    
    if(x < linesLength) {
        char aline[10000]={"\0"};


	char* arr[100];
        for(int i =0; i<100;i++){
        	arr[i] = '\0';
        }
        my_strcpy(aline, linesIn[x]);
        //:make sure that aline actually has
        // a length and is not passed a pointer

        int wrdCntPerThread = 0;

        bool inWord = false; //

        int i = -1;//first index-> first char in sentence
        int wLen = 0;// length of word
        int start = 0;//start of word

        do {
            i++;
            if (IsLetter(aline[i])) {
                //aline[i] = tolower(aline[i]);
                if (!inWord) {
                    start = i;//reached the start of a new word
                    inWord = true;
                }//end 2nd if
                wLen++;
            }//end 1st if
            else if (inWord) {
                char newWord[50] = {'\0'};
                //hipMemset(newWord, '\0', sizeof(newWord));//og: memset(...)
                //strncpy(newWord, &aline[start], wLen);//copies a word at the start of the new word of length wLen
                for(int j = 0; j < 50; j++){
                    int k = start;
                    if(k != start+wLen){
                        newWord[j] = aline[k];
                        k++;
                    }
                }
                if (wLen != 1 || newWord[0] == 'a' || newWord[0] == 'A' ||
                    newWord[0] == 'i' || newWord[0] == 'I') {
                    arr[wrdCntPerThread]=newWord;//
                    wrdCntPerThread++;
                }//end if
                wLen = 0;
                inWord = false;
                start = 0;
            }//end else if
        } while (aline[i] != '\n' && aline[i] != '\0');//end of while


        lock();
        for(int i = 0; i < wrdCntPerThread; i++){
            arrOut[globalIdx] = arr[i];
            globalIdx++;
        }
        unlock();

    }
}


//method call to parse the line
int parseLine( char** arr, int numCols, char** arrOut) {
    int dimx = numCols;
    int dimy = 10000;//10,000 or 100,000?
    dim3 grid, block;
    int num_bytes = dimx*sizeof(char*);

    char **d_a;
    hipMalloc((void **) &d_a, num_bytes);
    for(int i = 0; i < dimx; i++){
        hipMalloc((void**)&d_a[i], dimy* sizeof(char));
    }

    char **d_words;
    hipMalloc((void**)&d_words, 30000*sizeof(char*));
    for(int i = 0; i < 30000; i++){
        hipMalloc((void**)&d_a[i], 50* sizeof(char));
    }

    if (d_a == 0) {
        printf("error");
        return 0;
    }

    if(d_words == 0){
        printf("err device arr for words");
        return 0;
    }
    printf("fault2? \n");
    //hipMemset(d_a, '\0', dimy);
    hipMemset2D(d_a,dimx,'\0',dimx,dimy);
    //hipMemcpy(d_a, arr, dimy, hipMemcpyHostToDevice);
    hipMemcpy2D(d_a,dimx ,arr ,dimx ,dimx ,10000 , hipMemcpyHostToDevice);
    printf("fault3? \n");
    //hipMemset(d_words,'\0', 50);
    hipMemset2D(d_words,30000,'\0',30000,50);

    block.x = 4;
    block.y = 4;
    grid.x = ceil((float) dimx / block.x);
    grid.y = ceil((float) dimy / block.y);
    printf("fault3? \n");
    parselineKernel<<<grid, block>>>(d_a, d_words , numCols);
    printf("fault4? \n");
    //hipMemcpy(arrOut, d_words, 50, hipMemcpyDeviceToHost);
    hipMemcpy2D(arrOut,30000 ,d_words ,30000 ,30000 ,50 , hipMemcpyDeviceToHost);
    hipFree(d_words);
    hipFree(d_a);
    return 0;

}

